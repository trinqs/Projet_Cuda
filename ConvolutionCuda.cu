#include "hip/hip_runtime.h"
#include "Convolution.h"

#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>
#include <cstring>

using namespace std;
using namespace cv;

using ui32 = unsigned int;



struct matriceConvolution {
    int*matrice;
    int cols;
    int rows;
    int sommeCoefficients;
    int facteurMax;

    __host__ __device__ matriceConvolution(int* _matrice,int tailleMatrice): matrice(_matrice) ,cols(tailleMatrice), rows(tailleMatrice){

        //this ->matrice = _matrice.data();
        this->sommeCoefficients = 0;
        int sommeNegative = 0;
        int sommePositive = 0;
        for (int i=0; i<rows; i++){
            for (int j=0; j< cols; j++){
                this->sommeCoefficients += _matrice[i*cols+j];
                if (_matrice[i*cols+j] < 0){
                    sommeNegative +=_matrice[i*cols+j];
                }else{
                    sommePositive += _matrice[i*cols+j];
                }
            }
        }
        this->facteurMax = max(sommePositive,(sommeNegative*-1));
    }

    __device__ __host__  int getCols(){ return cols;}
    __device__ __host__ int getRows(){ return rows;}
    __device__ __host__ int getSommeCoefficients(){ return sommeCoefficients;}
    __device__ __host__ int getFacteurMax(){ return facteurMax;}
    __device__ __host__ int* getMatrice(){ return matrice;};

};


__device__ unsigned char calculPixel(int x, int y, // le thread,
                                     size_t imgCols, size_t imgRows, // taille de l'image
                                     int limCols, int limRows, // la taille du noyau
                                     int couleur, // quelle couche de pixel
                                     unsigned char* rgb, matriceConvolution noyau){ // le tableau des pixel de l'image, la matrice de convolution
    auto sum=0;
    printf(" x :%d , y: %d \n", x, y);
    printf(" couleur :%d  \n", couleur);

    for (int decalageCol = -limCols; decalageCol < limCols+1; decalageCol++){
        for (int decalageRow = -limRows; decalageRow < limRows+1; decalageRow++){

            //coefficient de la matrice de convolution à l'indice associé, on fait la rotation en même temps par le calcul d'indice
            sum += rgb[3*(( x + decalageRow )*imgCols+( y + decalageCol ))+couleur] * noyau.getMatrice()[ (decalageRow + limRows) *noyau.getCols()+ decalageCol + limCols ];
            //sum += rgb[1];
        }
    }
    //normalisation en dehors de la boucle pour faire moins d'arrondis
    if (noyau.getSommeCoefficients()==noyau.getFacteurMax()){
        sum/= noyau.getFacteurMax();
    }

    if (sum < 0){
        sum=0;
    } else if(sum >255){
        sum=255;
    }
    return sum;
}

__global__ void pasAlpha(unsigned char* rgb, unsigned char* g, size_t imgCols,size_t imgRow, matriceConvolution noyau){
    int limCols = noyau.getCols()/2;
    int limRows = noyau.getRows()/2;


    //int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    //int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    int tidx = blockIdx.y;
    int tidy = threadIdx.y;



    // si c'est pas un bord
    if( tidy >= limCols && tidy< imgCols-limCols && tidx >= limRows && tidx < imgRow-limRows){
        for( int i=0; i<3; i++){
            printf(" i :%d  \n", i);
            //g[3*(tidy*imgCols+tidx)+i] = calculPixel(tidx,tidy,imgCols,imgRow,limCols,limRows,i,rgb,noyau);
            g[3*(tidy*imgCols+tidx)+i] = rgb[3*(tidy*imgCols+tidx)+i];
            //g[2]=1;
        }
    }
    else{
        //std::cout<<"id thread x"+ tidx+", y "+tidy<<std::endl;
        for(int i= 0; i<3;i++){

            //g[3*((tidx)*imgCols+tidy)+i] = 255;
            g[3*((tidx)*imgCols+tidy)+i] = rgb[3*(tidy*imgCols+tidx)+i];

        }
    }
}

int main(int n, char* params[])
{
    Mat m_in;
    if (n==2 || n==3){
        m_in = cv::imread(params[1], IMREAD_UNCHANGED );
    }else{
        m_in = cv::imread("in.jpeg", IMREAD_UNCHANGED );
    }

    auto bgr = m_in.data; // c'est pas du rgb c'est du bgr

    int cols = m_in.cols;
    int rows = m_in.rows;
    auto sizeBgr = 3*(cols*rows);

    auto type = m_in.type();

    std::vector<unsigned char > g(cols*rows);



    unsigned char * bgr_d;
    unsigned char * g_d;

    vector<string> convolutionList = {"blur3","blur5","blur11","gaussianBlur3", "nettete3", "detectEdges3"};
    hipMalloc(&bgr_d, sizeBgr);
    hipMalloc(&g_d, cols*rows);

    hipMemcpy(bgr_d,bgr,sizeBgr, hipMemcpyHostToDevice);


    int nbThreadMaxParBloc = 1024;
    //dim3 block( 32, 4 );
    //dim3 grid( (cols-1)/block.y+1,(rows-1)/block.x+1 );
    dim3 block(1,rows);
    dim3 grid(1,cols);

    for (int i=0; i< convolutionList.size(); i++){
        if (convolutionList[i]==("blur3")){

            int tailleNoyaux = 3;
            vector<int> matrice({1,1,1,
                                 1,1,1,
                                 1,1,1});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);


            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);
            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_cu_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_cu_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_cu_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }

        /*}else if (convolutionList[i]==("blur5")){

            int tailleNoyaux = 5;
            vector<int> matrice({1,1,1,1,1,
                                 1,1,1,1,1,
                                 1,1,1,1,1,
                                 1,1,1,1,1,
                                 1,1,1,1,1});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);


            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);

            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }


        }else if (convolutionList[i]==("blur11")){

            int tailleNoyaux = 11;
            vector<int> matrice({1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1,
                                 1,1,1,1,1,1,1,1,1,1,1});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);

            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);

            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }


        }else if (convolutionList[i]==("gaussianBlur3")){

            int tailleNoyaux = 3;
            vector<int> matrice({1,2,1,
                                 2,4,2,
                                 1,2,1});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);

            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);

            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }

        }else if (convolutionList[i]==("nettete3")){

            int tailleNoyaux = 3;
            vector<int> matrice({0,-1,0,
                                 -1,5,-1,
                                 0,-1,0});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);


            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);

            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }
        }else if (convolutionList[i]==("detectEdges3")){

            int tailleNoyaux = 3;
            vector<int> matrice({-1,-1,-1,
                                 -1,8,-1,
                                 -1,-1,-1});

            matriceConvolution noyau = matriceConvolution(matrice.data(),tailleNoyaux);

            if(sizeBgr%3==0){
                pasAlpha<<<block,grid>>>( bgr_d, g_d, cols,rows, noyau);

            }
            if(sizeBgr%4==0){
                //de l'alpha
            }

            cv::Mat m_out( rows, cols, type, g.data() );
            hipMemcpy(g.data(),g_d,cols*rows,hipMemcpyDeviceToHost);
            if (n==3){
                string res = "out_" + convolutionList[i] + "_";
                res.append(params[2]);
                cv::imwrite( res, m_out );
            }else if(n==2){
                string res = "out_" +  convolutionList[i] + "_";
                res.append(params[1]);
                cv::imwrite( res, m_out );
            }else{
                string res = "out_" + convolutionList[i];
                res.append(".jpeg");
                cv::imwrite( res, m_out );
            }*/
        }
    }
    hipFree(bgr_d);
    hipFree(g_d);


    return 0;
}